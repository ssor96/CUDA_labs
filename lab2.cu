#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cstring>

texture<uchar4, 2, hipReadModeElementType> tex;

#define CSC(call) do { \
	hipError_t res = call;	\
	if (res != hipSuccess) { \
		fprintf(stderr, "CUDA Error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(res)); \
		exit(1); \
	} \
} while (0)

#define max(a, b) (a > b? a: b)
#define min(a, b) (a > b? b: a)

#define calc(val) for (i = 0; i < 256; ++i) cnt[i] = 0; \
			for (int dx = max(-r, -x); dx <= r && x + dx < w; ++dx) { \
				for (int dy = max(-r, -y); dy <= r && y + dy < h; ++dy) { \
					cnt[tex2D(tex, x + dx, y + dy).val]++; \
				} \
			} \
			cur = 0; \
			for (i = 0; i < 255 && 2 * cur <= all; ++i) { \
				cur += cnt[i]; \
			} \
			dst[y * w + x].val = i;

__global__ void kernel(uchar4 *dst, int w, int h, int r) {
	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;
	int x, y;
	ushort cnt[256];
	ushort cur;
	for(x = blockDim.x * blockIdx.x + threadIdx.x; x < w; x += offsetx) {
		for(y = blockDim.y * blockIdx.y + threadIdx.y; y < h; y += offsety) {
			int i;
			ushort all = (min(x + r, w - 1) - max(x - r, 0) + 1) * (min(y + r, h - 1) - max(y - r, 0) + 1);
			calc(x);
			calc(y);
			calc(z);
			dst[y * w + x].w = tex2D(tex, x, y).w;
		}
	}
}

int main() {
	int w, h;
	size_t sz = 256;
	char* name = (char*) malloc(sizeof(char) * sz);
	name[getline(&name, &sz, stdin) - 1] = '\0';
	FILE *in = fopen(name, "rb");
	fread(&w, sizeof(int), 1 , in);
	fread(&h, sizeof(int), 1 , in);
	uchar4 *data = (uchar4*)malloc(sizeof(uchar4) * h * w);
	fread(data, sizeof(uchar4), h * w, in);
	fclose(in);

	name[getline(&name, &sz, stdin) - 1] = '\0';
	int r;
	scanf("%d", &r);
	hipArray *arr;
	hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
	hipMallocArray(&arr, &ch, w, h);
	hipMemcpyToArray(arr, 0, 0, data, sizeof(uchar4) * h * w, hipMemcpyHostToDevice);

	tex.addressMode[0] = hipAddressModeClamp;
	tex.addressMode[1] = hipAddressModeClamp;
	tex.channelDesc = ch;
	tex.filterMode = hipFilterModePoint;
	tex.normalized = false; 

	hipBindTextureToArray(tex, arr, ch);
	uchar4 *dev_data;
	hipMalloc(&dev_data, sizeof(uchar4) * h * w);


	hipEvent_t start, stop;
	float t;
	CSC(hipEventCreate(&start));
	CSC(hipEventCreate(&stop));

	CSC(hipEventRecord(start, 0));
	
	kernel<<<dim3(8, 8), dim3(8, 8)>>>(dev_data, w, h, r);
	CSC(hipGetLastError());
	
	CSC(hipEventRecord(stop, 0));
	CSC(hipEventSynchronize(stop));
	CSC(hipEventElapsedTime(&t, start, stop));	
	printf("time = %f\n", t);
	CSC(hipEventDestroy(start));
	CSC(hipEventDestroy(stop));

	hipMemcpy(data, dev_data, sizeof(uchar4) * h * w, hipMemcpyDeviceToHost);
	FILE *out = fopen(name, "wb");
	fwrite(&w, sizeof(int), 1, out);
	fwrite(&h, sizeof(int), 1, out);
	fwrite(data, sizeof(uchar4), w * h, out);
	fclose(out);

	hipUnbindTexture(tex);
	hipFreeArray(arr);
	hipFree(dev_data);
	free(data);
	free(name);
	return 0;
}